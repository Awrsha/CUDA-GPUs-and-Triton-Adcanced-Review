
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vector_add(const float *a, const float *b, float *c, int N) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    const int N = 512;
    float a[N], b[N], c[N];

    // Initialize vectors
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i;
    }

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel with N/256 blocks and 256 threads per block
    vector_add<<<N / 256, 256>>>(d_a, d_b, d_c, N);

    hipMemcpy(c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    // Print result
    for (int i = 0; i < N; i++) {
        std::cout << c[i] << " ";
    }

    // Clean up
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}